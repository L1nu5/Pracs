#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void square(int *l, int *m, int *n)
{
	int id = threadIdx.x;
	__shared__ int add, mul;

	if (id == 1)
	{
		add = (l[0] + m[0]);
	}
	else
	{
		mul = m[0] * m[0];
	}

	__syncthreads();
	n[2] = add * 10 + mul;
}

int main()
{
	int num[1], diff[1];
	int *a, *b, *c;
	int squ[3];

	cout << "Enter number : " << endl;
	cin >> num[0];

	diff[0] = num[0] - 10;

	hipMalloc(&a, sizeof(int));
	hipMalloc(&b, sizeof(int));
	hipMalloc(&c, sizeof(int)*3);

	hipMemcpy(a, num,sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b, diff,sizeof(int), hipMemcpyHostToDevice);

	square << <1, 2 >> >(a, b, c);

	hipMemcpy(squ, c, sizeof(int)* 3, hipMemcpyDeviceToHost);

	cout << endl << "Square is: ";
	cout << squ[2];

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}
